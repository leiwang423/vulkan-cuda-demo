#include <hip/hip_runtime.h>

// CUDA kernel
__global__ void computeKernel(float* data)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    data[idx] = data[idx] * data[idx];
}

// CUDA kernel launcher
void launchComputeKernel(float* d_data, int dataSize)
{
    int blockSize = 256;
    int numBlocks = (dataSize + blockSize - 1) / blockSize;
    computeKernel<<<numBlocks, blockSize>>>(d_data);
    hipDeviceSynchronize();
}
